
#include <hip/hip_runtime.h>
#include <stdio.h>

#define X 10
#define Y 10

__global__ void add_mat_kernel(int *a, int *b, int *c)
{
    int i;
    int bi = blockIdx.x;
    int ti = threadIdx.x;

    printf("%d:%d\n", bi, ti);

    i = threadIdx.x + blockIdx.x * blockDim.x;
    c[i] = a[i] + b[i];
}

int main(void)
{
    int i, j;
    int size = X * Y *  sizeof(int);

    int cpu_a[X][Y];
    int cpu_b[X][Y];
    int cpu_c[X][Y];

    int *gpu_a = NULL;
    int *gpu_b = NULL;
    int *gpu_c = NULL;

    for (i = 0; i < X; i++) {
        for (j = 0; j < Y; j++) {
            cpu_a[i][j] = 1;
            cpu_b[i][j] = 2;
        }
    }

    hipMalloc((void **)&gpu_a, size);
    hipMalloc((void **)&gpu_b, size);
    hipMalloc((void **)&gpu_c, size);

    hipMemcpy(gpu_a, cpu_a, size, hipMemcpyHostToDevice);
    hipMemcpy(gpu_b, cpu_b, size, hipMemcpyHostToDevice);

    add_mat_kernel<<<20, 5>>>(gpu_a, gpu_b, gpu_c);

    hipDeviceSynchronize();

    hipMemcpy(cpu_c, gpu_c, size, hipMemcpyDeviceToHost);

    for (i = 0; i < X; i++) {
        for (j = 0; j < Y; j++) {
            printf("%d ", cpu_c[i][j]);
        }
        printf("\n");
    }
    hipFree(gpu_a);
    hipFree(gpu_b);
    hipFree(gpu_c);

    return 0;
}
