
#include <hip/hip_runtime.h>
#include <stdio.h>

#define X 10
#define Y 10

__global__ void add_mat_kernel(int *a, int *b, int *c)
{
    int x, y, i;

    x = blockIdx.x * blockDim.x + threadIdx.x;
    y = blockIdx.y * blockDim.y + threadIdx.y;

//    printf("%d:%d\n", x, y);

    i = y * gridDim.x * blockDim.x + x;

    c[i] = a[i] + b[i];
}

int main(void)
{
    int i, j;
    int size = X * Y *  sizeof(int);

    int cpu_a[X][Y];
    int cpu_b[X][Y];
    int cpu_c[X][Y];

    int *gpu_a = NULL;
    int *gpu_b = NULL;
    int *gpu_c = NULL;

    for (i = 0; i < X; i++) {
        for (j = 0; j < Y; j++) {
            cpu_a[i][j] = i * X + j;
            cpu_b[i][j] = -(i * X + j);
        }
    }

    for (i = 0; i < X; i++) {
        for (j = 0; j < Y; j++) {
            printf("%2d ", cpu_a[i][j]);
        }
        printf("\n");
    }

    for (i = 0; i < X; i++) {
        for (j = 0; j < Y; j++) {
            printf("%2d ", cpu_b[i][j]);
        }
        printf("\n");
    }

    hipMalloc((void **)&gpu_a, size);
    hipMalloc((void **)&gpu_b, size);
    hipMalloc((void **)&gpu_c, size);

    hipMemcpy(gpu_a, cpu_a, size, hipMemcpyHostToDevice);
    hipMemcpy(gpu_b, cpu_b, size, hipMemcpyHostToDevice);

    dim3 dimBlock(5, 2);
    dim3 dimGrid(2, 5);

    add_mat_kernel<<<dimGrid, dimBlock>>>(gpu_a, gpu_b, gpu_c);

    hipDeviceSynchronize();

    hipMemcpy(cpu_c, gpu_c, size, hipMemcpyDeviceToHost);

    for (i = 0; i < X; i++) {
        for (j = 0; j < Y; j++) {
            printf("%2d ", cpu_c[i][j]);
        }
        printf("\n");
    }
    hipFree(gpu_a);
    hipFree(gpu_b);
    hipFree(gpu_c);

    return 0;
}
