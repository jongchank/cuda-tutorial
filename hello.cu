
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_cuda()
{
    printf("Hello from GPU\n");
}

int main(void)
{
    printf("Hello from CPU\n");

    hello_cuda<<<1, 3>>>();

    hipDeviceSynchronize();

    return 0;
}
