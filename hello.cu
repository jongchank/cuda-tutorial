
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_cuda()
{
    printf("Hello from GPU (%d, %d)\n", blockIdx.x, threadIdx.x);
}

int main(void)
{
    printf("Hello from CPU\n");

    hello_cuda<<<2, 3>>>();

    hipDeviceSynchronize();

    return 0;
}
