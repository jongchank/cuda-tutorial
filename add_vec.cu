
#include <hip/hip_runtime.h>
#include <stdio.h>

#define LEN 10

__global__ void add_vec_kernel(int *a, int *b, int *c)
{
    int i;
    int bi = blockIdx.x;
    int ti = threadIdx.x;

    printf("%d:%d\n", bi, ti);

    i = threadIdx.x + blockIdx.x * blockDim.x;
    c[i] = a[i] + b[i];
}

int main(void)
{
    int i;
    int size = LEN * sizeof(int);

    int cpu_a[LEN] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10};
    int cpu_b[LEN] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10};
    int cpu_c[LEN] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

    int *gpu_a = NULL;
    int *gpu_b = NULL;
    int *gpu_c = NULL;

    hipMalloc((void **)&gpu_a, size);
    hipMalloc((void **)&gpu_b, size);
    hipMalloc((void **)&gpu_c, size);

    hipMemcpy(gpu_a, cpu_a, size, hipMemcpyHostToDevice);
    hipMemcpy(gpu_b, cpu_b, size, hipMemcpyHostToDevice);

    add_vec_kernel<<<2, 5>>>(gpu_a, gpu_b, gpu_c);

    hipDeviceSynchronize();

    hipMemcpy(cpu_c, gpu_c, size, hipMemcpyDeviceToHost);

    for (i = 0; i < LEN; i++) {
        printf("C[%d] = %d\n", i, cpu_c[i]);
    }

    hipFree(gpu_a);
    hipFree(gpu_b);
    hipFree(gpu_c);

    return 0;
}
